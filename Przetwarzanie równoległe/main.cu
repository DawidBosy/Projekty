
#include <iostream>
#include <string.h>
#include <time.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

__host__ __device__ bool isSubsetSum(int* set, int n, int sum)
{
   
    if (sum == 0)
        return true;
    if (n == 0)
        return false;
 
    if (set[n - 1] > sum)
        return isSubsetSum(set, n - 1, sum);
 
    return isSubsetSum(set, n - 1, sum)
           || isSubsetSum(set, n - 1, sum - set[n - 1]);
}

__global__ void CudaRun(int iterations, int arrayLength,int* numbers,float * result){
    int x,i,n;
    hiprandState state;
    hiprand_init(1234,0,0,&state);
    
    clock_t startTime = clock();
    for(i = 0;i<iterations;i++){      
        for(x = 0;x<arrayLength;x++){
            numbers[x] = (int)(ceil((hiprand_uniform(&state)*(50 + 1))) - 1);
        }
        n = sizeof(numbers) / sizeof(numbers[0]);
        isSubsetSum(numbers,n,(int)(ceil((hiprand_uniform(&state)*(50 + 1))) - 1)+50);
    }
    
    clock_t endTime = clock();
    *result = (float)(endTime - startTime);
}

int main(){
    
    int problemNum = 6000;
    int arrayLength = 50;
    float result;
    srand(time(NULL));
    int startTime = time(NULL);

    int *numbers;
    int *d_numbers;
    float * d_result;

    // Allocate host memory
    numbers   = (int*)malloc(sizeof(int) * arrayLength);

    hipMalloc((void**)&d_numbers, sizeof(int)*arrayLength);
    hipMalloc((void**)&d_result,sizeof(float));



    // Transfer data from host to device memory
    hipMemcpy(d_numbers, &numbers, sizeof(int)*arrayLength, hipMemcpyHostToDevice);

    // Executing kernel 
    CudaRun<<<1,1>>>(problemNum,arrayLength,d_numbers,d_result);
    
    // Transfer data back to host memory
    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    printf("Seconds passed in sequence run: %f \n",(double)result/(double)CLOCKS_PER_SEC);

    // Deallocate device memory
    hipFree(d_result);
    hipFree(d_numbers);

    // Deallocate host memory
    free(numbers); 
}
